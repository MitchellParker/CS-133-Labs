#include "hip/hip_runtime.h"
#include "lib/cnn.cuh"
#include "cnn_gpu.cuh"

__shared__ float in[4][36][36];
// __shared__ float wt[4][5][5];

__global__ void cnn_gpu(
    float* input,
    float* weight,
    float* bias,
    float* output)
{

  int ii = threadIdx.x;
  int hh = threadIdx.y * 2;
  int ww = threadIdx.z * 2;

  int i =   blockIdx.x * blockDim.x + threadIdx.x;
  int h = ( blockIdx.y * blockDim.y + threadIdx.y ) * 2;
  int w = ( blockIdx.z * blockDim.z + threadIdx.z ) * 2;


  float c[2][2];

  // Bias
  c[0][0] = c[0][1] = c[1][0] = c[1][1] = bias[i];

  // Convolution
  for (int j = 0; j < kNum; j++) {
    // cache inputs
    in[ii][hh  ][ww  ] = input(j,h  ,w  );
    in[ii][hh  ][ww+1] = input(j,h  ,w+1);
    in[ii][hh+1][ww  ] = input(j,h+1,w  );
    in[ii][hh+1][ww+1] = input(j,h+1,w+1);
    if (hh < 4 && ww >= 2*blockDim.z-4) {
      in[ii][hh  ][4+ww  ] = input(j,h  ,4+w  );
      in[ii][hh  ][4+ww+1] = input(j,h  ,4+w+1);
      in[ii][hh+1][4+ww  ] = input(j,h+1,4+w  );
      in[ii][hh+1][4+ww+1] = input(j,h+1,4+w+1);
    }
    if (hh >= 2*blockDim.y-4 && ww < 4) {
      in[ii][4+hh  ][ww  ] = input(j,4+h  ,w  );
      in[ii][4+hh  ][ww+1] = input(j,4+h  ,w+1);
      in[ii][4+hh+1][ww  ] = input(j,4+h+1,w  );
      in[ii][4+hh+1][ww+1] = input(j,4+h+1,w+1);
    }
    if (hh >= 2*blockDim.y-4 || ww >= 2*blockDim.z-4) {
      in[ii][4+hh  ][4+ww  ] = input(j,4+h  ,4+w  );
      in[ii][4+hh  ][4+ww+1] = input(j,4+h  ,4+w+1);
      in[ii][4+hh+1][4+ww  ] = input(j,4+h+1,4+w  );
      in[ii][4+hh+1][4+ww+1] = input(j,4+h+1,4+w+1);
    }
    // if (hh < kKernel && ww < kKernel) {
    //   wt[ii][hh][ww] = weight(i,j,hh,ww);
    // }
    __syncthreads();

    for (int q = 0; q < kKernel; q++) {
      for (int p = 0; p < kKernel; p++) {
        // float temp = wt[ii][p][q];
        float temp = weight(i,j,p,q);
        c[0][0] += temp * in[ii][hh  +p][ww  +q];
        c[0][1] += temp * in[ii][hh  +p][ww+1+q];
        c[1][0] += temp * in[ii][hh+1+p][ww  +q];
        c[1][1] += temp * in[ii][hh+1+p][ww+1+q];
      }
    }
    __syncthreads();
  }

  // Max pooling then ReLU
  output(i,h/2,w/2) = max(0.f, max(
    max(c[0][0], c[0][1]),
    max(c[1][0], c[1][1])
  ));

}